#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
This example demonstrates how to use the Cuda OpenGL bindings to
dynamically modify a vertex buffer using a Cuda kernel.

The steps are:
1. Create an empty vertex buffer object (VBO)
2. Register the VBO with Cuda
3. Map the VBO for writing from Cuda
4. Run Cuda kernel to modify the vertex positions
5. Unmap the VBO
6. Render the results using OpenGL

Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_math_constants.h>

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

texture<unsigned char, 3, hipReadModeNormalizedFloat> tex;  // 3D texture

hipArray *d_volumeArray = 0;

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
	return (a + (b - 1)) / b;
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void position_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	//// calculate simple sine wave pattern
	//float freq = 4.0f;
	//float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
	float voxel = tex3D(tex, u, v, 0.0f);

	// write output vertex
	pos[y*width + x] = make_float4(u, voxel, v, 1.0f);
}

__global__ void normal_vbo_kernel(float4 *pos, float4 *norms,unsigned int width, unsigned int height)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// write output normal
	//norms[y*width + x] = make_float4(u, voxel, v, 1.0f);
}

extern "C"
void calculate_position_kernel(float4 *pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	position_vbo_kernel << < grid, block >> >(pos, mesh_width, mesh_height, time);
}

extern "C"
void calculate_normal_kernel(float4 *pos, float4 *norms, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	normal_vbo_kernel << < grid, block >> >(pos, norms, mesh_width, mesh_height);
}

extern "C"
void initCuda(const unsigned char *h_volume, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void *)h_volume, volumeSize.width * sizeof(unsigned char), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
	tex.addressMode[1] = hipAddressModeWrap;
	tex.addressMode[2] = hipAddressModeWrap;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}